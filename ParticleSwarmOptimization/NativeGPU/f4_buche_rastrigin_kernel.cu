#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "bbob_generators.cuh"

__constant__ double d_OMEGA = 0.64;
__constant__ double d_phi = 1.4;

__constant__ double PI = 3.1415;

__device__ double fitness_function(double x[], int number_of_variables)
{
    double tmp = 0., tmp2 = 0.;
    int i;
    double result;

    result = 0.0;
    for(i = 0; i < number_of_variables; ++i)
    {
        tmp += cos(2 * 3.1415 * x[i]);
        tmp2 += x[i] * x[i];
    }
    result = 10.0 * ((double)(long)number_of_variables - tmp) + tmp2 + 0;
    return result;
}

__device__ double wrapped_fitness_function(double x[], int number_of_variables,
                                           double* xopt, double fopt, double penalty)
{
    transform_vars_shift(x, number_of_variables, xopt);
    transform_vars_oscillate(x, number_of_variables);
    transform_vars_brs(x, number_of_variables);
    double temp[1];
    temp[0] = fitness_function(x, number_of_variables);
    transform_obj_shift(temp, 1, fopt);
    transform_obj_penalize(temp, 1, penalty);

    return temp[0];
}


extern "C" {
    __global__ void generateData(int dimension,
                                 int rseed,
                                 int function,
                                 int instance,
                                 double* vars_shift_xopt,
                                 double* obj_shift_fopt)
    {
        bbob2009_compute_xopt(vars_shift_xopt, rseed, dimension);

        for(int i = 0; i < dimension; i += 2)
            vars_shift_xopt[i] = fabs(vars_shift_xopt[i]);
        
        obj_shift_fopt[0] = bbob2009_compute_fopt(function, instance);
    }

    __global__ void kernelUpdateParticle(double *positions, double *velocities,
                                         double *pBests, double *gBest,
                                         int particlesCount, int dimensionsCount,
                                         double r1, double r2)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount * dimensionsCount)
            return;

        velocities[i] = d_OMEGA * velocities[i] + r1 * (pBests[i] - positions[i])
            + r2 * (gBest[i % dimensionsCount] - positions[i]);

        // Update posisi particle
        positions[i] += velocities[i];
    }

    __global__ void kernelUpdatePBest(double *positions, double *pBests, double* gBest,
                                      int particlesCount, int dimensionsCount,
                                      double* xopt, double fopt, double penalty)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        double tempParticle1[MAX_DIMENSIONS];
        double tempParticle2[MAX_DIMENSIONS];

        if(i >= particlesCount * dimensionsCount || i % dimensionsCount != 0)
            return;

        for(int j = 0; j < dimensionsCount; j++)
        {
            tempParticle1[j] = positions[i + j];
            tempParticle2[j] = pBests[i + j];
        }

        if(wrapped_fitness_function(tempParticle1, dimensionsCount, xopt, fopt, penalty) <
           wrapped_fitness_function(tempParticle2, dimensionsCount, xopt, fopt, penalty))
        {
            for(int k = 0; k < dimensionsCount; k++)
                pBests[i + k] = positions[i + k];
        }
    }
}