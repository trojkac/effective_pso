#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "bbob_generators.cuh"

__constant__ double d_OMEGA = 0.64;
__constant__ double d_phi = 1.4;

__constant__ double PI = 3.1415;

__device__ double fitness_function(double x[], int number_of_variables)
{
    size_t i = 0;
    double result;
    double s1 = 0.0, s2 = 0.0, tmp;

    for(i = 0; i < number_of_variables - 1; ++i)
    {
        tmp = (x[i] * x[i] - x[i + 1]);
        s1 += tmp * tmp;
        tmp = (x[i] - 1.0);
        s2 += tmp * tmp;
    }
    result = 100.0 * s1 + s2;

    return result;
}

__device__ double wrapped_fitness_function(double x[], int number_of_variables,
                                           double* M, double* b, double fopt)
{
    transform_vars_affine(x, number_of_variables, M, b);
    double temp[1];
    temp[0] = fitness_function(x, number_of_variables);
    transform_obj_shift(temp, 1, fopt);

    return temp[0];
}


extern "C" {
    __global__ void generateData(int dimension,
                                 int rseed,
                                 int function,
                                 int instance,
                                 double* vars_affine_m,
                                 double* vars_affine_b,
                                 double* obj_shift_fopt)
    {
        obj_shift_fopt[0] = bbob2009_compute_fopt(function, instance);

        double rot1[MAX_DIMENSIONS][MAX_DIMENSIONS];
        double rot2[MAX_DIMENSIONS][MAX_DIMENSIONS];

        bbob2009_compute_rotation(dimension, rot1, rseed + 1000000);
        bbob2009_compute_rotation(dimension, rot2, rseed);

        double *current_row;

        double factor = coco_double_max(1.0, sqrt((double)dimension) / 8.0);

        /* Compute affine transformation */
        for(int row = 0; row < dimension; ++row)
        {
            current_row = vars_affine_m + row * dimension;
            for(int column = 0; column < dimension; ++column)
            {
                current_row[column] = rot1[row][column];
                if(row == column)
                    current_row[column] *= factor;
            }
            vars_affine_b[row] = 0.5;
        }
    }

    __global__ void transposeKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int particlesCount,
        int dimensionsCount,
        double* M, double* b, double fopt)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleLoc[i] += particleVel[i];
        }

        clamp(particleLoc, dimensionsCount, -5.0, 5.0);

        double newValue = wrapped_fitness_function(particleLoc, dimensionsCount, M, b, fopt);

        if(newValue < personalBestValues[i])
        {
            personalBestValues[i] = newValue;

            double* particlePersonalBest = personalBests + i * dimensionsCount;

            for(int i = 0; i < dimensionsCount; i++)
                particlePersonalBest[i] = particleLoc[i];
        }
    }

    __global__ void updateVelocityKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int* neighbors,
        int particlesCount,
        int dimensionsCount,
        double phi1,
        double phi2)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;
        double* particleBest = personalBests + i * dimensionsCount;
        double particleBestValue = personalBestValues[i];

        int* particleNeighbors = neighbors + i * 2;

        int leftNeighborId = particleNeighbors[0];
        double* leftNeighborBest = personalBests + leftNeighborId * dimensionsCount;
        double leftNeighborBestVal = personalBestValues[leftNeighborId];

        int rightNeighborId = particleNeighbors[1];
        double* rightNeighborBest = personalBests + rightNeighborId * dimensionsCount;
        double rightNeighborBestVal = personalBestValues[rightNeighborId];

        double* globalBest = particleBest;
        double globalBestVal = particleBestValue;

        if(leftNeighborBestVal < globalBestVal)
        {
            globalBest = leftNeighborBest;
            globalBestVal = leftNeighborBestVal;
        }

        if(rightNeighborBestVal < globalBestVal)
        {
            globalBest = rightNeighborBest;
        }

        double toPersonalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, particleBest, dimensionsCount, toPersonalBest);

        double toGlobalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, globalBest, dimensionsCount, toGlobalBest);

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleVel[i] = particleVel[i] * d_OMEGA + phi1 * toGlobalBest[i] + phi2 * toPersonalBest[i];
        }
    }
}