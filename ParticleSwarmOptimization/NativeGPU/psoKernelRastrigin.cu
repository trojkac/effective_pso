#include <hip/hip_runtime.h>

#include <math_functions.h>

const int NUM_OF_DIMENSIONS = 3;

__constant__ double d_OMEGA= 0.64;
__constant__ double d_phi = 1.4;

__constant__ double PI = 3.1415;

__device__ double tempParticle1[NUM_OF_DIMENSIONS];
__device__ double tempParticle2[NUM_OF_DIMENSIONS];

// Rosenbrock function
__device__ double fitness_function(double x[], int dimensionsCount)
{
	int A = 10;
	double result = 0.0;

	for (int i = 0; i < dimensionsCount; i++)
	{
	    result += x[i] * x[i] - A * cos(2 * PI * x[i]);
	}

    return A * dimensionsCount + result;
}

extern "C" {
	__global__ void kernelUpdateParticle(double *positions, double *velocities, 
										 double *pBests, double *gBest,
										 int particlesCount, int dimensionsCount,
										 double r1, double r2)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
    
		if(i >= particlesCount * dimensionsCount)
			return;

		velocities[i] = d_OMEGA * velocities[i] + r1 * (pBests[i] - positions[i])
				+ r2 * (gBest[i % dimensionsCount] - positions[i]);

		// Update posisi particle
		positions[i] += velocities[i];
	}

	__global__ void kernelUpdatePBest(double *positions, double *pBests, double* gBest,
									  int particlesCount, int dimensionsCount)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
    
		if(i >= particlesCount * dimensionsCount || i % dimensionsCount != 0)
			return;

		for (int j = 0; j < dimensionsCount; j++)
		{
			tempParticle1[j] = positions[i + j];
			tempParticle2[j] = pBests[i + j];
		}

		if (fitness_function(tempParticle1, dimensionsCount) < fitness_function(tempParticle2, dimensionsCount))
		{
			for (int k = 0; k < dimensionsCount; k++)
				pBests[i + k] = positions[i + k];
		}
	}
}