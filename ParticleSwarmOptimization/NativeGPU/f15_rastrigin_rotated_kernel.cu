#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "bbob_generators.cuh"




__device__ double fitness_function(double x[], int number_of_variables)
{
    size_t i = 0;
    double result;
    double sum1 = 0.0, sum2 = 0.0;

    for(i = 0; i < number_of_variables; ++i)
    {
        sum1 += cos(coco_two_pi * x[i]);
        sum2 += x[i] * x[i];
    }
    result = 10.0 * ((double)(long)number_of_variables - sum1) + sum2;

    return result;
}

__device__ double wrapped_fitness_function(double x[], int number_of_variables,
                                           double* xopt, double fopt, double asymmetric, double* M, double* b)
{
    transform_vars_shift(x, number_of_variables, xopt);
    transform_vars_affine(x, number_of_variables, M, b);
    transform_obj_oscillate(x, number_of_variables);
    transform_vars_asymmetric(x, number_of_variables, asymmetric);
    transform_vars_affine(x, number_of_variables, M, b);
    double temp[1];
    temp[0] = fitness_function(x, number_of_variables);
    transform_obj_shift(temp, 1, fopt);

    return temp[0];
}


extern "C" {
    __global__ void generateData(int dimension,
                                 int rseed,
                                 int function,
                                 int instance,
                                 double* M,
                                 double* b,
                                 double* vars_shift_xopt,
                                 double* obj_shift_fopt)
    {
        bbob2009_compute_xopt(vars_shift_xopt, rseed, dimension);
        obj_shift_fopt[0] = bbob2009_compute_fopt(function, instance);


        double rot1[MAX_DIMENSIONS][MAX_DIMENSIONS];
        double rot2[MAX_DIMENSIONS][MAX_DIMENSIONS];

        bbob2009_compute_rotation(dimension, rot1, rseed + 1000000);
        bbob2009_compute_rotation(dimension, rot2, rseed);

        double *current_row;

        for(int i = 0; i < dimension; ++i)
        {
            b[i] = 0.0;
            current_row = M + i * dimension;
            for(int j = 0; j < dimension; ++j)
            {
                current_row[j] = 0.0;
                for(int k = 0; k < dimension; ++k)
                {
                    double exponent = 1.0 * (int)k / ((double)(long)dimension - 1.0);
                    current_row[j] += rot1[i][k] * pow(sqrt(10.0), exponent) * rot2[k][j];
                }
            }
        }
    }

    __global__ void transposeKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int particlesCount,
        int dimensionsCount,
        double* xopt, double fopt, double asymmetric, double* M, double* b)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleLoc[i] += particleVel[i];
        }

        clamp(particleLoc, dimensionsCount, -5.0, 5.0);

        double tempLocation[MAX_DIMENSIONS];

        for(int i = 0; i < dimensionsCount; i++)
        {
            tempLocation[i] = particleLoc[i];
        }

        double newValue = wrapped_fitness_function(tempLocation, dimensionsCount, xopt, fopt, asymmetric, M, b);

        if(newValue < personalBestValues[i])
        {
            personalBestValues[i] = newValue;

            double* particlePersonalBest = personalBests + i * dimensionsCount;

            for(int i = 0; i < dimensionsCount; i++)
                particlePersonalBest[i] = particleLoc[i];
        }
    }
}