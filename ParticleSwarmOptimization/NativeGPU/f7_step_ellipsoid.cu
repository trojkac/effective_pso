#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "bbob_generators.cuh"

__constant__ double d_OMEGA = 0.64;
__constant__ double d_phi = 1.4;

__constant__ double PI = 3.1415;

__device__ double fitness_function(double x[], int number_of_variables, double* xopt, double fopt, double *rot1, double *rot2)
{
    const double condition = 100;
    const double alpha = 10.0;
    size_t i, j;
    double penalty = 0.0, x1;
    double result;

    double tempx[MAX_DIMENSIONS];
    double tempxx[MAX_DIMENSIONS];

    double* current_row;

    for(i = 0; i < number_of_variables; ++i)
    {
        double tmp;
        tmp = fabs(x[i]) - 5.0;
        if(tmp > 0.0)
            penalty += tmp * tmp;
    }

    for(i = 0; i < number_of_variables; ++i)
    {
        double c1;
        tempx[i] = 0.0;
        current_row = rot2 + i * number_of_variables;
        c1 = sqrt(pow(condition / 10., (double)i / (double)(number_of_variables - 1)));
        for(j = 0; j < number_of_variables; ++j)
        {
            tempx[i] += c1 * current_row[j] * (x[j] - xopt[j]);
        }
    }
    x1 = tempx[0];

    for(i = 0; i < number_of_variables; ++i)
    {
        if(fabs(tempx[i]) > 0.5)
            tempx[i] = coco_double_round(tempx[i]);
        else
            tempx[i] = coco_double_round(alpha * tempx[i]) / alpha;
    }

    for(i = 0; i < number_of_variables; ++i)
    {
        tempxx[i] = 0.0;
        current_row = rot1 + i * number_of_variables;
        for(j = 0; j < number_of_variables; ++j)
        {
            tempxx[i] += current_row[j] * tempx[j];
        }
    }

    /* Computation core */
    result = 0.0;
    for(i = 0; i < number_of_variables; ++i)
    {
        double exponent;
        exponent = (double)(long)i / ((double)(long)number_of_variables - 1.0);
        result += pow(condition, exponent) * tempxx[i] * tempxx[i];
        ;
    }
    result = 0.1 * coco_double_max(fabs(x1) * 1.0e-4, result) + penalty + fopt;

    return result;
}

__device__ double wrapped_fitness_function(double x[], int number_of_variables,
                                           double* xopt, double* rot1, double* rot2, double fopt)
{
    double temp[1];
    temp[0] = fitness_function(x, number_of_variables, xopt, fopt, rot1, rot2);
    return temp[0];
}


extern "C" {
    __global__ void generateData(int dimension,
                                 int rseed,
                                 int function,
                                 int instance,
                                 double* rot1,
                                 double* rot2,
                                 double* vars_shift_xopt,
                                 double* obj_shift_fopt)
    {
        bbob2009_compute_xopt(vars_shift_xopt, rseed, dimension);
        obj_shift_fopt[0] = bbob2009_compute_fopt(function, instance);

        double rot1d[MAX_DIMENSIONS][MAX_DIMENSIONS];
        double rot2d[MAX_DIMENSIONS][MAX_DIMENSIONS];

        bbob2009_compute_rotation(dimension, rot1d, rseed + 1000000);
        bbob2009_compute_rotation(dimension, rot2d, rseed);

        double *current_row_1;
        double *current_row_2;

        for(int i = 0; i < dimension; ++i)
        {
            current_row_1 = rot1 + i * dimension;
            current_row_2 = rot2 + i * dimension;

            for(int j = 0; j < dimension; ++j)
            {
                current_row_1[j] = rot1d[i][j];
                current_row_2[j] = rot2d[i][j];
            }
        }
    }


    __global__ void transposeKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int particlesCount,
        int dimensionsCount,
        double* xopt, double* rot1, double* rot2, double fopt)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleLoc[i] += particleVel[i];
        }

        clamp(particleLoc, dimensionsCount, -5.0, 5.0);

        double newValue = wrapped_fitness_function(particleLoc, dimensionsCount, xopt, rot1, rot2, fopt);

        if(newValue < personalBestValues[i])
        {
            personalBestValues[i] = newValue;

            double* particlePersonalBest = personalBests + i * dimensionsCount;

            for(int i = 0; i < dimensionsCount; i++)
                particlePersonalBest[i] = particleLoc[i];
        }
    }

    __global__ void updateVelocityKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int* neighbors,
        int particlesCount,
        int dimensionsCount,
        double phi1,
        double phi2)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;
        double* particleBest = personalBests + i * dimensionsCount;
        double particleBestValue = personalBestValues[i];

        int* particleNeighbors = neighbors + i * 2;

        int leftNeighborId = particleNeighbors[0];
        double* leftNeighborBest = personalBests + leftNeighborId * dimensionsCount;
        double leftNeighborBestVal = personalBestValues[leftNeighborId];

        int rightNeighborId = particleNeighbors[1];
        double* rightNeighborBest = personalBests + rightNeighborId * dimensionsCount;
        double rightNeighborBestVal = personalBestValues[rightNeighborId];

        double* globalBest = particleBest;
        double globalBestVal = particleBestValue;

        if(leftNeighborBestVal < globalBestVal)
        {
            globalBest = leftNeighborBest;
            globalBestVal = leftNeighborBestVal;
        }

        if(rightNeighborBestVal < globalBestVal)
        {
            globalBest = rightNeighborBest;
        }

        double toPersonalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, particleBest, dimensionsCount, toPersonalBest);

        double toGlobalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, globalBest, dimensionsCount, toGlobalBest);

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleVel[i] = particleVel[i] * d_OMEGA + phi1 * toGlobalBest[i] + phi2 * toPersonalBest[i];
        }
    }
}