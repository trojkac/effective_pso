#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "bbob_generators.cuh"

__constant__ double d_OMEGA = 0.64;
__constant__ double d_phi = 1.4;

__constant__ double PI = 3.1415;

__device__ double fitness_function(double x[], int number_of_variables, double* rotation, double number_of_peaks, double* peak_values, double* x_local, double* arr_scales)
{
    size_t i, j; /* Loop over dim */
    double tmx[MAX_DIMENSIONS];
    double a = 0.1;
    double tmp2, f = 0., f_add, tmp, f_pen = 0., f_true = 0.;
    double fac;
    double result;
    double* row;
    double* row2;

    fac = -0.5 / (double)number_of_variables;

    /* Boundary handling */
    for(i = 0; i < number_of_variables; ++i)
    {
        tmp = fabs(x[i]) - 5.;
        if(tmp > 0.)
        {
            f_pen += tmp * tmp;
        }
    }
    f_add = f_pen;
    /* Transformation in search space */
    /* TODO: this should rather be done in f_gallagher */

    for(i = 0; i < number_of_variables; i++)
    {
        tmx[i] = 0;
        row = rotation + i * number_of_variables;
        for(j = 0; j < number_of_variables; ++j)
        {
            tmx[i] += row[j] * x[j];
        }
    }
    /* Computation core*/
    for(i = 0; i < number_of_peaks; ++i)
    {
        row = arr_scales + i * (int)number_of_peaks;
        tmp2 = 0.;
        for(j = 0; j < number_of_variables; ++j)
        {
            row2 = x_local + j * number_of_variables;
            tmp = (tmx[j] - row2[i]);
            tmp2 += row[j] * tmp * tmp;
        }
        tmp2 = peak_values[i] * exp(fac * tmp2);
        f = coco_double_max(f, tmp2);
    }

    f = 10. - f;
    if(f > 0)
    {
        f_true = log(f) / a;
        f_true = pow(exp(f_true + 0.49 * (sin(f_true) + sin(0.79 * f_true))), a);
    }
    else if(f < 0)
    {
        f_true = log(-f) / a;
        f_true = -pow(exp(f_true + 0.49 * (sin(0.55 * f_true) + sin(0.31 * f_true))), a);
    }
    else
        f_true = f;

    f_true *= f_true;
    f_true += f_add;
    result = f_true;
    return result;
}

__device__ double wrapped_fitness_function(double x[], int number_of_variables,
                                           double* rotation, double number_of_peaks, double* peak_values, double* x_local, double* arr_scales)
{
    double temp[1];
    temp[0] = fitness_function(x, number_of_variables, rotation, number_of_peaks, peak_values, x_local, arr_scales);

    return temp[0];
}


extern "C" {
    __global__ void generateData(int dimension,
                                 int rseed,
                                 double* rotation, 
                                 double number_of_peaks, 
                                 double* peak_values, 
                                 double* x_local, 
                                 double* arr_scales)
    {
        size_t i, j, k;
        double maxcondition = 1000.0;
        double maxcondition1 = 1000.0;
        double b, c;
        double random_numbers[101 * MAX_DIMENSIONS];
        double fitvalues[2] = { 1.1, 9.1 };

        if(number_of_peaks == 101.0)
        {
            maxcondition1 = sqrt(maxcondition1);
            b = 10.;
            c = 5.;
        }
        else if(number_of_peaks == 21.0)
        {
            b = 9.8;
            c = 4.9;
        }

        double rot[MAX_DIMENSIONS][MAX_DIMENSIONS];

        bbob2009_compute_rotation(dimension, rot, rseed);

        double* row;

        for(i = 0; i < dimension; i++)
        {
            row = rotation + i * dimension;
            for(j = 0; j < dimension; i++)
            {
                row[j] = rot[i][j];
            }
        }

        double arrCondition[101];
        arrCondition[0] = maxcondition1;
        peak_values[0] = 10;

        for(i = 1; i < number_of_peaks; ++i)
        {
            arrCondition[i] = pow(maxcondition, (double)(i) / ((double)(number_of_peaks - 2)));
            peak_values[i] = (double)(i - 1) / (double)(number_of_peaks - 2) * (fitvalues[1] - fitvalues[0])
                + fitvalues[0];
        }

        for(i = 0; i < number_of_peaks; ++i)
        {
            row = arr_scales + i * (int)number_of_peaks;
            for(j = 0; j < dimension; ++j)
            {
                row[j] = pow(arrCondition[i],
                                             (j / ((double)(dimension - 1)) - 0.5));
            }
        }

        bbob2009_unif(random_numbers, dimension * number_of_peaks, rseed);
        for(i = 0; i < dimension; ++i)
        {
            row = x_local + i * dimension;
            double* rotrow = rotation + i * dimension;
            for(j = 0; j < number_of_peaks; ++j)
            {
                row[j] = 0.;
                for(k = 0; k < dimension; ++k)
                {
                    row[j] += rotrow[k] * (b * random_numbers[j * dimension + k] - c);
                }
                if(j == 0)
                {
                    row[j] *= 0.8;
                }
            }
        }
    }

    __global__ void transposeKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int particlesCount,
        int dimensionsCount,
        double* rotation, double number_of_peaks, double* peak_values, double* x_local, double* arr_scales)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleLoc[i] += particleVel[i];
        }

        clamp(particleLoc, dimensionsCount, -5.0, 5.0);

        double newValue = wrapped_fitness_function(particleLoc, dimensionsCount, rotation, number_of_peaks, peak_values, x_local, arr_scales);

        if(newValue < personalBestValues[i])
        {
            personalBestValues[i] = newValue;

            double* particlePersonalBest = personalBests + i * dimensionsCount;

            for(int i = 0; i < dimensionsCount; i++)
                particlePersonalBest[i] = particleLoc[i];
        }
    }

    __global__ void updateVelocityKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int* neighbors,
        int particlesCount,
        int dimensionsCount,
        double phi1,
        double phi2)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;
        double* particleBest = personalBests + i * dimensionsCount;
        double particleBestValue = personalBestValues[i];

        int* particleNeighbors = neighbors + i * 2;

        int leftNeighborId = particleNeighbors[0];
        double* leftNeighborBest = personalBests + leftNeighborId * dimensionsCount;
        double leftNeighborBestVal = personalBestValues[leftNeighborId];

        int rightNeighborId = particleNeighbors[1];
        double* rightNeighborBest = personalBests + rightNeighborId * dimensionsCount;
        double rightNeighborBestVal = personalBestValues[rightNeighborId];

        double* globalBest = particleBest;
        double globalBestVal = particleBestValue;

        if(leftNeighborBestVal < globalBestVal)
        {
            globalBest = leftNeighborBest;
            globalBestVal = leftNeighborBestVal;
        }

        if(rightNeighborBestVal < globalBestVal)
        {
            globalBest = rightNeighborBest;
        }

        double toPersonalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, particleBest, dimensionsCount, toPersonalBest);

        double toGlobalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, globalBest, dimensionsCount, toGlobalBest);

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleVel[i] = particleVel[i] * d_OMEGA + phi1 * toGlobalBest[i] + phi2 * toPersonalBest[i];
        }
    }
}