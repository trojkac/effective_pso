#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "bbob_generators.cuh"

__constant__ double d_OMEGA = 0.64;
__constant__ double d_phi = 1.4;

__device__ double fitness_function(double x[], int number_of_variables, double* rotation, int number_of_peaks, double* peak_values, double* x_local, double* arr_scales)
{
    size_t i, j; /* Loop over dim */
    double tmx[MAX_DIMENSIONS];
    double a = 0.1;
    double tmp2, f = 0., f_add, tmp, f_pen = 0., f_true = 0.;
    double fac;
    double result;
    double* row;
    double* row2;

    fac = -0.5 / (double)number_of_variables;

    /* Boundary handling */
    for(i = 0; i < number_of_variables; ++i)
    {
        tmp = fabs(x[i]) - 5.;
        if(tmp > 0.)
        {
            f_pen += tmp * tmp;
        }
    }
    f_add = f_pen;
    /* Transformation in search space */
    /* TODO: this should rather be done in f_gallagher */

    for(i = 0; i < number_of_variables; i++)
    {
        tmx[i] = 0;
        row = rotation + i * number_of_variables;
        for(j = 0; j < number_of_variables; ++j)
        {
            tmx[i] += row[j] * x[j];
        }
    }
    /* Computation core*/
    for(i = 0; i < number_of_peaks; ++i)
    {
        row = arr_scales + i * (int)number_of_peaks;
        tmp2 = 0.;
        for(j = 0; j < number_of_variables; ++j)
        {
            row2 = x_local + j * number_of_variables;
            tmp = (tmx[j] - row2[i]);
            tmp2 += row[j] * tmp * tmp;
        }
        tmp2 = peak_values[i] * exp(fac * tmp2);
        f = coco_double_max(f, tmp2);
    }

    f = 10. - f;
    if(f > 0)
    {
        f_true = log(f) / a;
        f_true = pow(exp(f_true + 0.49 * (sin(f_true) + sin(0.79 * f_true))), a);
    }
    else if(f < 0)
    {
        f_true = log(-f) / a;
        f_true = -pow(exp(f_true + 0.49 * (sin(0.55 * f_true) + sin(0.31 * f_true))), a);
    }
    else
        f_true = f;

    f_true *= f_true;
    f_true += f_add;
    result = f_true;
    return result;
}

__device__ double wrapped_fitness_function(double x[], int number_of_variables,
                                           double* rotation, int number_of_peaks, double* peak_values, double* x_local, double* arr_scales)
{
    double temp[1];
    temp[0] = fitness_function(x, number_of_variables, rotation, number_of_peaks, peak_values, x_local, arr_scales);

    return temp[0];
}


extern "C" {
    __global__ void generateData(int dimension,
                                 int rseed,
                                 double* rotation, 
                                 int number_of_peaks, 
                                 double* peak_values, 
                                 double* x_local, 
                                 double* arr_scales)
    {

    }

    __global__ void transposeKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int particlesCount,
        int dimensionsCount,
        double* rotation, int number_of_peaks, double* peak_values, double* x_local, double* arr_scales)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleLoc[i] += particleVel[i];
        }

        clamp(particleLoc, dimensionsCount, -5.0, 5.0);

        double tempLocation[MAX_DIMENSIONS];

        for(int i = 0; i < dimensionsCount; i++)
        {
            tempLocation[i] = particleLoc[i];
        }

        double newValue = wrapped_fitness_function(tempLocation, dimensionsCount, rotation, number_of_peaks, peak_values, x_local, arr_scales);

        if(newValue < personalBestValues[i])
        {
            personalBestValues[i] = newValue;

            double* particlePersonalBest = personalBests + i * dimensionsCount;

            for(int i = 0; i < dimensionsCount; i++)
                particlePersonalBest[i] = particleLoc[i];
        }
    }

    __global__ void updateVelocityKernel(
        double* positions,
        double* velocities,
        double* personalBests,
        double* personalBestValues,
        int* neighbors,
        int particlesCount,
        int dimensionsCount,
        double phi1,
        double phi2)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= particlesCount) return;

        double* particleLoc = positions + i * dimensionsCount;
        double* particleVel = velocities + i * dimensionsCount;
        double* particleBest = personalBests + i * dimensionsCount;
        double particleBestValue = personalBestValues[i];

        int* particleNeighbors = neighbors + i * 2;

        int leftNeighborId = particleNeighbors[0];
        double* leftNeighborBest = personalBests + leftNeighborId * dimensionsCount;
        double leftNeighborBestVal = personalBestValues[leftNeighborId];

        int rightNeighborId = particleNeighbors[1];
        double* rightNeighborBest = personalBests + rightNeighborId * dimensionsCount;
        double rightNeighborBestVal = personalBestValues[rightNeighborId];

        double* globalBest = particleBest;
        double globalBestVal = particleBestValue;

        if(leftNeighborBestVal < globalBestVal)
        {
            globalBest = leftNeighborBest;
            globalBestVal = leftNeighborBestVal;
        }

        if(rightNeighborBestVal < globalBestVal)
        {
            globalBest = rightNeighborBest;
        }

        double toPersonalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, particleBest, dimensionsCount, toPersonalBest);

        double toGlobalBest[MAX_DIMENSIONS];
        vector_between(particleLoc, globalBest, dimensionsCount, toGlobalBest);

        for(int i = 0; i < dimensionsCount; i++)
        {
            particleVel[i] = particleVel[i] * d_OMEGA + phi1 * toGlobalBest[i] + phi2 * toPersonalBest[i];
        }
    }
}